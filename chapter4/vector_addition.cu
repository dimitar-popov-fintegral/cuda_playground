#include "hip/hip_runtime.h"
#include <stdlib.h>
#include "../common/book.h"

#define N 100


////////////////////////////////////////
__global__ void cuda_add(int * a, int * b, int * c){

  int tid = blockIdx.x;
  if(tid<N){
    
    a[tid] = b[tid] + c[tid];

  }

}


////////////////////////////////////////
void fill_vectors(int * a, int * b, int * c){

  for(int i=0; i<=N; i++){

    a[i] = 0;
    b[i] = i*i;
    c[i] = -i;

  }

}


////////////////////////////////////////
int main(void){

  int a[N], b[N], c[N];
  int * deva, * devb, * devc;

  // allocating ON-DEVICE memory using hipMalloc(...)
  HANDLE_ERROR( hipMalloc( (void**)&deva, N * sizeof(int) ) );
  HANDLE_ERROR( hipMalloc( (void**)&devb, N * sizeof(int) ) );
  HANDLE_ERROR( hipMalloc( (void**)&devc, N * sizeof(int) ) );

  // fill vectors with numbers
  int * pa = &a[0];
  int * pb = &b[0];
  int * pc = &c[0];
  fill_vectors(pa, pb, pc);

  // copy HOST -> DEVICE using hipMemcpy(...)
  HANDLE_ERROR(hipMemcpy(devb, b, N * sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(devc, c, N * sizeof(int), hipMemcpyHostToDevice));

  // perform DEVICE operation
  cuda_add<<<N,1>>>(deva, devb, devc);

  // return result by copying from DEVICE -> HOST
  HANDLE_ERROR(hipMemcpy(a, deva, N * sizeof(int), hipMemcpyDeviceToHost));

  // print final results
  for(int i=0; i<N; i++){

    printf("[%d] + [%d] = [%d] \n", b[i], c[i], a[i]);

  }

  hipFree(deva);
  hipFree(devb);
  hipFree(devc);

  return 0;

}
